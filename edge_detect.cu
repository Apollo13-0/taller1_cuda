#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define SOBEL_FILTER_WIDTH 3


// Define the Sobel kernel for x and y directions
__constant__ int SobelX[SOBEL_FILTER_WIDTH][SOBEL_FILTER_WIDTH] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int SobelY[SOBEL_FILTER_WIDTH][SOBEL_FILTER_WIDTH] = {
    {-1, 2, 1},
    { 0, 0, 0},
    { 1, 2, 1}
};

// This kernel computes the Sobel filter on the input image
__global__ void sobelFilterKernel(unsigned char *inputImage, unsigned char *outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Ensure threads are within the image bounds
    if (x >= width || y >= height) {
        return;
    }

    float dx = 0.0f, dy = 0.0f;

    // Apply the Sobel filter
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* inputImage[(y-1)*width + (x-1)]) + (-2*inputImage[y*width+(x-1)]) + (-1*inputImage[(y+1)*width+(x-1)]) +
             (    inputImage[(y-1)*width + (x+1)]) + ( 2*inputImage[y*width+(x+1)]) + (   inputImage[(y+1)*width+(x+1)]);
        dy = (    inputImage[(y-1)*width + (x-1)]) + ( 2*inputImage[(y-1)*width+x]) + (   inputImage[(y-1)*width+(x+1)]) +
             (-1* inputImage[(y+1)*width + (x-1)]) + (-2*inputImage[(y+1)*width+x]) + (-1*inputImage[(y+1)*width+(x+1)]);

        // Compute the gradient magnitude
        float edgeVal = sqrtf(dx * dx + dy * dy);

        // Normalize and set the output pixel
        outputImage[y * width + x] = (unsigned char)min(max(edgeVal, 0.0f), 255.0f);
    }
}

int main(int argc, char* argv[]) {
    int width, height, channels;

    if (argc < 2) {
        printf("Error: Please provide the image filename as a parameter!");
        return -1;
    }

    const char* imageName = argv[1];
    
    // Load the image using stb_image
    unsigned char* h_inputImage = stbi_load(imageName, &width, &height, &channels, 1); // Load as grayscale
    if (!h_inputImage) {
        printf("Error: Could not load the image!");
        return -1;
    }

    printf("Image loaded: %ix%i, channels: %x\n", width, height, channels);

    clock_t start = clock();

    // Allocate host memory for the output image
    unsigned char* h_outputImage = (unsigned char*)malloc(width * height * sizeof(unsigned char));

    // Allocate device memory
    unsigned char* d_inputImage;
    unsigned char* d_outputImage;
    hipMalloc((void**)&d_inputImage, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_outputImage, width * height * sizeof(unsigned char));

    // Copy input image from host to device
    hipMemcpy(d_inputImage, h_inputImage, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Launch the Sobel kernel
    sobelFilterKernel<<<gridDim, blockDim>>>(d_inputImage, d_outputImage, width, height);

    clock_t end = clock();

    double time = (double)(end-start)/CLOCKS_PER_SEC;

    printf("GPU time = %fs\n", time);

    // Copy the result back to host
    hipMemcpy(h_outputImage, d_outputImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save the Sobel result to a file (optional)
    stbi_write_png("sobel_output.png", width, height, 1, h_outputImage, width);

    // Free device and host memory
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    free(h_inputImage);
    free(h_outputImage);

    return 0;
}
