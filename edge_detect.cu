#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "stb_image.h"   // For loading images
#include "stb_image_write.h" // For saving images
//#include <opencv2/opencv.hpp> // For displaying images

#define SOBEL_FILTER_WIDTH 3

//using namespace cv;

// Define the Sobel kernel for x and y directions
__constant__ int SobelX[SOBEL_FILTER_WIDTH][SOBEL_FILTER_WIDTH] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int SobelY[SOBEL_FILTER_WIDTH][SOBEL_FILTER_WIDTH] = {
    {-1, 2, 1},
    { 0, 0, 0},
    { 1, 2, 1}
};

// CUDA kernel to apply the Sobel operator
__global__ void sobelFilterKernel(unsigned char *inputImage, unsigned char *outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Ensure threads are within the image bounds
    if (x >= width || y >= height) {
        return;
    }

    float Gx = 0.0f, Gy = 0.0f;

    // Apply the Sobel filter
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int imgX = min(max(x + i, 0), width - 1);  // Ensure indices are within bounds
            int imgY = min(max(y + j, 0), height - 1); // Ensure indices are within bounds

            unsigned char pixelValue = inputImage[imgY * width + imgX];

            Gx += pixelValue * SobelX[i + 1][j + 1];
            Gy += pixelValue * SobelY[i + 1][j + 1];
        }
    }

    // Compute the gradient magnitude
    float edgeVal = sqrtf(Gx * Gx + Gy * Gy);

    // Normalize and set the output pixel
    outputImage[y * width + x] = (unsigned char)min(max(edgeVal, 0.0f), 255.0f);
}

int main() {
    int width, height, channels;
    
    // Load the image using stb_image
    unsigned char* h_inputImage = stbi_load("img/image1.png", &width, &height, &channels, 1); // Load as grayscale
    if (!h_inputImage) {
        std::cerr << "Error: Could not load the image!" << std::endl;
        return -1;
    }

    std::cout << "Image loaded: " << width << "x" << height << ", channels: " << channels << std::endl;

    // Display the image using OpenCV
    Mat inputImage(height, width, CV_8UC1, h_inputImage); // Create an OpenCV Mat from the image data
    imshow("Original Image", inputImage);  // Show the original image
    waitKey(0); // Wait for a key press

    // Allocate host memory for the output image
    unsigned char* h_outputImage = (unsigned char*)malloc(width * height * sizeof(unsigned char));

    // Allocate device memory
    unsigned char* d_inputImage;
    unsigned char* d_outputImage;
    hipMalloc((void**)&d_inputImage, width * height * sizeof(unsigned char));
    hipMalloc((void**)&d_outputImage, width * height * sizeof(unsigned char));

    // Copy input image from host to device
    hipMemcpy(d_inputImage, h_inputImage, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Launch the Sobel kernel (make sure the Sobel kernel is defined earlier)
    sobelFilterKernel<<<gridDim, blockDim>>>(d_inputImage, d_outputImage, width, height);

    // Copy the result back to host
    hipMemcpy(h_outputImage, d_outputImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Create an OpenCV Mat for the output and display it
    //Mat outputImage(height, width, CV_8UC1, h_outputImage);
    //imshow("Sobel Edge Detection", outputImage); // Show the Sobel edge-detected image
    //waitKey(0); // Wait for a key press

    // Save the Sobel result to a file (optional)
    stbi_write_png("sobel_output.png", width, height, 1, h_outputImage, width);

    // Free device and host memory
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    free(h_inputImage);
    free(h_outputImage);

    return 0;
}
