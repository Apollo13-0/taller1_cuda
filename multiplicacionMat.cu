// multiplication of two matrices 4x4
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3

__global__ void matrix_mul(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    if (i < n && j < n) {
        for (int k = 0; k < n; k++)
            sum += a[i * n + k] * b[k * n + j];
        c[i * n + j] = sum;
    }
}

int main() {
    int n = N;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            a[i * n + j] = i + j;
            b[i * n + j] = i * j;
        }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockSize(N, N);
    dim3 gridSize((n + N - 1) / N, (n + N - 1) / N);
    matrix_mul<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%d ", c[i * n + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}